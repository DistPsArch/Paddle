
#include <hip/hip_runtime.h>
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#if defined _WIN32 || defined __APPLE__
#else
#define _LINUX
#endif
#if defined(PADDLE_WITH_CUDA) && defined(PADDLE_WITH_HETERPS)

#include "paddle/fluid/framework/data_feed.h"

namespace paddle {
namespace framework {

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

// CUDA: use 512 threads per block
const int CUDA_NUM_THREADS = 512;
// CUDA: number of blocks for threads.
inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}
// fill slot values
__global__ void FillSlotValueOffsetKernel(
    const int ins_num, const int used_slot_num, size_t *slot_value_offsets,
    const int *uint64_offsets, const int uint64_slot_size,
    const int *float_offsets, const int float_slot_size,
    const UsedSlotGpuType *used_slots) {

  int col_num = ins_num + 1;
  int uint64_cols = uint64_slot_size + 1;
  int float_cols = float_slot_size + 1;

  CUDA_KERNEL_LOOP(slot_idx, used_slot_num) {
    int value_off = slot_idx * col_num;
    slot_value_offsets[value_off] = 0;
    auto &info = used_slots[slot_idx];
    if (info.is_uint64_value) {
      for (int k = 0; k < ins_num; ++k) {
        int pos = k * uint64_cols + info.slot_value_idx;
        int num = uint64_offsets[pos + 1] - uint64_offsets[pos];
        PADDLE_ENFORCE(num >= 0, "The number of slot size must be ge 0.");
        slot_value_offsets[value_off + k + 1] =
            slot_value_offsets[value_off + k] + num;
      }
    } else {
      for (int k = 0; k < ins_num; ++k) {
        int pos = k * float_cols + info.slot_value_idx;
        int num = float_offsets[pos + 1] - float_offsets[pos];
        PADDLE_ENFORCE(num >= 0, "The number of slot size must be ge 0.");
        slot_value_offsets[value_off + k + 1] =
            slot_value_offsets[value_off + k] + num;
      }
    }
  }
}

  // 填充gpu_slot_offset
  // uint64_use_slot_size_是uint64的slot数量
  // float_use_slot_size_是float的slot数量
  // value.d_uint64_offset存储每个ins的uint64 slot lod信息
  // value.d_float_offset存储每个ins的float slot lod信息
  // 比如 ins 10个，uint64 2个，float slot 3个
  // 那么 value.d_uint64_offset的shape就是10 * (2 + 1)
  // 那么 value.d_float_offset的shape就是10 * (3 + 1)
  // used_slot_gpu_types 每个slot的信息,包括是否为uint64, 以及slot_value_idx
  // 这个函数就是填充slot_value_offsets
  //
void SlotRecordInMemoryDataFeed::FillSlotValueOffset(
    const int ins_num, const int used_slot_num, size_t *slot_value_offsets,
    const int *uint64_offsets, const int uint64_slot_size,
    const int *float_offsets, const int float_slot_size,
    const UsedSlotGpuType *used_slots,
    cudaStream_t stream) {
  FillSlotValueOffsetKernel<<<GET_BLOCKS(used_slot_num), CUDA_NUM_THREADS, 0,
                              stream>>>(
      ins_num, used_slot_num, slot_value_offsets, uint64_offsets,
      uint64_slot_size, float_offsets, float_slot_size, used_slots);
  cudaStreamSynchronize(stream);
}

  // uint64_feas保存的是所有样本的uint64 key
  // uint64_ins_lens shape (ins_num + 1), 保存每个ins的uint64 feasign num数量
  // uint64_offset shape(ins_num * (uint64_slot_num + 1)),保存每个样本的uint64_slot_offset 
__global__ void CopyForTensorKernel(
    const int used_slot_num, const int ins_num, void **dest,
    const size_t *slot_value_offsets, const uint64_t *uint64_feas,
    const int *uint64_offsets, const int *uint64_ins_lens,
    const int uint64_slot_size, const float *float_feas,
    const int *float_offsets, const int *float_ins_lens,
    const int float_slot_size, const UsedSlotGpuType *used_slots) {
  int col_num = ins_num + 1;
  int uint64_cols = uint64_slot_size + 1;
  int float_cols = float_slot_size + 1;
  CUDA_KERNEL_LOOP(i, ins_num * used_slot_num) {
    int slot_idx = i / ins_num;
    int ins_idx = i % ins_num;
    uint32_t value_offset = slot_value_offsets[slot_idx * col_num + ins_idx];
    auto &info = used_slots[slot_idx];
    if (info.is_uint64_value) {
      uint64_t *up = reinterpret_cast<uint64_t *>(dest[slot_idx]);
      int index = info.slot_value_idx + uint64_cols * ins_idx;
      int old_off = uint64_offsets[index];
      int num = uint64_offsets[index + 1] - old_off;
      PADDLE_ENFORCE(num >= 0, "The number of slot size must be ge 0.");
      int uint64_value_offset = uint64_ins_lens[ins_idx];
      for (int k = 0; k < num; ++k) {
        up[k + value_offset] = uint64_feas[k + old_off + uint64_value_offset];
      }
    } else {
      float *fp = reinterpret_cast<float *>(dest[slot_idx]);
      int index = info.slot_value_idx + float_cols * ins_idx;
      int old_off = float_offsets[index];
      int num = float_offsets[index + 1] - old_off;
      PADDLE_ENFORCE(num >= 0, "The number of slot size must be ge 0.");
      int float_value_offset = float_ins_lens[ins_idx];
      for (int k = 0; k < num; ++k) {
        fp[k + value_offset] = float_feas[k + old_off + float_value_offset];
      }
    }
  }
}

  // pack->resize_gpu_slot_offsets(slot_total_num * sizeof(size_t));
  // gpu_slot_offset的shape是 use_slot_size * (ins_num + 1)
  // d_uint64_keys保存的是所有样本的uint64 key
  // d_uint64_lens shape (ins_num + 1), 保存每个ins的uint64 feasign num数量
  // d_uint64_offset shape(ins_num * (uint64_slot_num + 1)),保存每个样本的uint64_slot_offset 
void SlotRecordInMemoryDataFeed::CopyForTensor(
    const int ins_num, const int used_slot_num, void **dest,
    const size_t *slot_value_offsets, const uint64_t *uint64_feas,
    const int *uint64_offsets, const int *uint64_ins_lens,
    const int uint64_slot_size, const float *float_feas,
    const int *float_offsets, const int *float_ins_lens,
    const int float_slot_size, const UsedSlotGpuType *used_slots,
    cudaStream_t stream) {
  CopyForTensorKernel<<<GET_BLOCKS(used_slot_num * ins_num), CUDA_NUM_THREADS,
                        0, stream>>>(
      used_slot_num, ins_num, dest, slot_value_offsets, uint64_feas,
      uint64_offsets, uint64_ins_lens, uint64_slot_size, float_feas,
      float_offsets, float_ins_lens, float_slot_size, used_slots);
  cudaStreamSynchronize(stream);
}

}  // namespace framework
}  // namespace paddle
#endif
